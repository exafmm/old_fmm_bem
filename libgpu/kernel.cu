#include "hip/hip_runtime.h"
#define P2P_KERNEL(p2p_kernel,p2p_kernel_core) \
extern "C" __global__ \
void p2p_kernel(int *nvec,float *ivec,float *jvec,float *kvec)\
{\
  int bx=blockIdx.x;\
  int tx=threadIdx.x;\
  int mblok=nvec[2];\
  int ib,jb,jbase,jsize,jblok,nij;\
  int tx7,jj7;\
  int i,j,ij,jj;\
  float dxij,dyij,dzij,rij,rsij=0;\
  float eps=1e-6;\
  float pi14=0.25/M_PI;\
  float veci[6],veck[4];\
  __shared__ float vecj[NBLOK0*7];\
  rij=rsij;\
  ib=bx*NBLOK0+tx;\
  for(i=0;i<6;i++) veci[i]=ivec[6*ib+i];\
  for(i=0;i<4;i++) veck[i]=0.0f;\
  tx7=tx*7;\
  nij=nvec[bx*mblok+10];\
  for(ij=0;ij<nij;ij++){\
    jbase=nvec[bx*mblok+2*ij+11];\
    jsize=nvec[bx*mblok+2*ij+12];\
    jblok=(jsize+NBLOK0-1)/NBLOK0;\
    for(j=0;j<jblok-1;j++){\
      jb=j*NBLOK0+jbase+tx;\
      for(i=0;i<7;i++) vecj[tx7+i]=jvec[7*jb+i];\
      __syncthreads();\
      for(jj=jj7=0;jj<NBLOK0;jj++){\
        p2p_kernel_core;\
      }\
      __syncthreads();\
    }\
    jb=j*NBLOK0+jbase+tx;\
    for(i=0;i<7;i++) vecj[tx7+i]=jvec[7*jb+i];\
    jb=j*NBLOK1+jbase;\
    __syncthreads();\
    for(jj=jj7=0;jj<jsize-(j*NBLOK0);jj++){\
      p2p_kernel_core;\
    }\
    __syncthreads();\
  }\
  for(i=0;i<4;i++) kvec[4*ib+i]=veck[i];\
}

#define P2M_KERNEL(p2m_kernel,p2m_kernel_core) \
extern "C" __global__ \
void p2m_kernel(int *nvec,float *ivec,float *jvec,float *kvec)\
{\
  int bx=blockIdx.x;\
  int tx=threadIdx.x;\
  int i,j,k,m,n,ib,jb,jj,jj7,jbase,jsize,jblok,nm,nms;\
  int mblok=nvec[2],mp=nvec[6],nb,nc[3],nd;\
  float rb=scal[0],xmin=scal[1],ymin=scal[2],zmin=scal[3];\
  float xjc,yjc,zjc,xjjc,yjjc,zjjc,rh,al,be,eps=1e-6;\
  float xx,s2,p,pn,p1,p2,fact,ere,eim,rhm,rhn;\
  __shared__ int mg[NBLOK1],ng[NBLOK1];\
  __shared__ float veci[2*NBLOK1],vecj[7*NBLOK1],veck[MPMAX];\
  __shared__ float bnm[MPMAX];\
  ib=bx*NBLOK1+tx;\
  for(i=0;i<NBLOK1;i++){\
    ng[i]=0;\
    mg[i]=0;\
  }\
  for(n=0;n<mp;n++){\
    for(m=0;m<=n;m++){\
      nms=n*(n+1)/2+m;\
      ng[nms]=n;\
      mg[nms]=m;\
    }\
  }\
  jblok=(MPMAX+NBLOK1-1)/NBLOK1;\
  for(j=0;j<jblok-1;j++){\
    jb=j*NBLOK1+tx;\
    veck[jb]=kvec[jb];\
    veck[jb]=kvec[jb];\
    __syncthreads();\
  }\
  if(j*NBLOK1+tx<MPMAX){\
    jb=j*NBLOK1+tx;\
    veck[jb]=kvec[jb];\
    veck[jb]=kvec[jb];\
  }\
  __syncthreads();\
  for(i=0;i<2;i++) veci[2*tx+i]=0;\
  __syncthreads();\
  nb=nvec[bx*mblok+10];\
  jbase=nvec[bx*mblok+11];\
  jsize=nvec[bx*mblok+12];\
  for(i=0;i<3;i++) nc[i]=0;\
  k=0;\
  i=1;\
  while(nb!=0){\
    j=2-k;\
    nc[j]=nc[j]+nb%2*i;\
    nb=nb/2;\
    j=k+1;\
    k=j%3;\
    if(k==0) i=i*2;\
  }\
  nd=nc[0];\
  nc[0]=nc[1];\
  nc[1]=nc[2];\
  nc[2]=nd;\
  xjc=xmin+(nc[0]+0.5)*rb;\
  yjc=ymin+(nc[1]+0.5)*rb;\
  zjc=zmin+(nc[2]+0.5)*rb;\
  jblok=(jsize+NBLOK1-1)/NBLOK1;\
  for(j=0;j<jblok-1;j++){\
    jb=j*NBLOK1+jbase+tx;\
    for(i=0;i<7;i++) vecj[7*tx+i]=jvec[7*jb+i];\
    __syncthreads();\
    for(jj=jj7=0;jj<NBLOK1;jj++){\
      p2m_kernel_core;\
      __syncthreads();\
    }\
  }\
  jb=j*NBLOK1+jbase+tx;\
  for(i=0;i<7;i++) vecj[7*tx+i]=jvec[7*jb+i];\
  __syncthreads();\
  for(jj=jj7=0;jj<jsize-(j*NBLOK1);jj++){\
    p2m_kernel_core;\
    __syncthreads();\
  }\
  for(i=0;i<2;i++) ivec[2*ib+i]=veci[2*tx+i];\
}

#define M2L_KERNEL(m2l_kernel,m2l_kernel_core) \
extern "C" __global__ \
void m2l_kernel(int *nvec,float *ivec,float *jvec,float *kvec,float *lvec)\
{\
  int bx=blockIdx.x;\
  int tx=threadIdx.x;\
  int i,j,k,m,n,ib,jb,ij,nij,jbase,jblok,je,nms,nmk,nks,jks,jnk;\
  int mblok=nvec[2],mp=nvec[6],nrbm=nvec[7];\
  int nb=0,nc[3],nd=0,mpdnm=(4*mp*mp*mp-mp)/3;\
  float rb=scal[0],eps=1e-6;\
  float xijc=0,yijc=0,zijc=0,rh,cnmre,cnmim,dnmre,dnmim;\
  float sr,ank,ajk,ajn,fnmm,fnpm;\
  float vecd[2];\
  __shared__ int mg[NBLOK1],ng[NBLOK1];\
  __shared__ float veci[6*NBLOK1],vecj[2*NBLOK1];\
  __shared__ float ynmre[MPMAX],ynmim[MPMAX];\
  nc[0]=0;\
  rh=xijc+yijc+zijc+eps+nb+nc[0]+nd;\
  ib=bx*NBLOK1+tx;\
  nij=nvec[bx*mblok+10];\
  for(i=0;i<NBLOK1;i++){\
    ng[i]=0;\
    mg[i]=0;\
  }\
  for(n=0;n<mp;n++){\
    for(m=0;m<=n;m++){\
      nms=n*(n+1)/2+m;\
      ng[nms]=n;\
      mg[nms]=m;\
    }\
  }\
  jblok=(MPMAX+NBLOK1-1)/NBLOK1;\
  for(j=0;j<jblok-1;j++){\
    jb=j*NBLOK1+tx;\
    ynmre[jb]=kvec[2*jb+0];\
    ynmim[jb]=kvec[2*jb+2];\
    __syncthreads();\
  }\
  if(j*NBLOK1+tx<MPMAX){\
    jb=j*NBLOK1+tx;\
    ynmre[jb]=kvec[2*jb+0];\
    ynmim[jb]=kvec[2*jb+1];\
  }\
  __syncthreads();\
  for(i=0;i<6;i++) veci[6*tx+i]=0;\
  __syncthreads();\
  for(ij=0;ij<nij;ij++){\
    jbase=nvec[bx*mblok+2*ij+11];\
    je=nvec[bx*mblok+2*ij+12];\
    jb=jbase+tx;\
    for(i=0;i<2;i++) vecj[2*tx+i]=jvec[2*jb+i];\
    __syncthreads();\
    m2l_kernel_core;\
    for(i=0;i<2;i++) veci[6*tx+3*i]+=vecd[i];\
    __syncthreads();\
  }\
  for(i=0;i<2;i++) ivec[2*ib+i]=veci[6*tx+3*i];\
}

#define L2P_KERNEL(l2p_kernel,l2p_kernel_core) \
extern "C" __global__ \
void l2p_kernel(int *nvec,float *ivec,float *jvec,float *kvec,float *lvec)\
{\
  int bx=blockIdx.x;\
  int tx=threadIdx.x;\
  int i,j,k,m,n,ib,jb,jbase,jblok,nm,nms;\
  int mblok=nvec[2],mp=nvec[6],nb,nc[3],nd;\
  float rb=scal[0],xmin=scal[1],ymin=scal[2],zmin=scal[3];\
  float xic,yic,zic,xiic,yiic,ziic,r,th,ph;\
  float xx,yy=0,s2,p,pn,p1,p2,fact,ere,eim,eps=1e-6;\
  float rsre=0,rsim=0,rrre=0,rrim=0,rthre=0,rthim=0,rphre=0,rphim=0;\
  float g=0,gr=0,gth=0,gph=0,gx=0,gy=0,gz=0;\
  float bnm,bth=0;\
  __shared__ float veci[6*NBLOK1],vecj[2*NBLOK1],veck[4*NBLOK1],vecl[MPMAX];\
  r=yy+bth+rsre+rsim+rrre+rrim+rthre+rthim+rphre+rphim+g+gr+gth+gph+gx+gy+gz;\
  ib=bx*NBLOK1+tx;\
  jblok=(MPMAX+NBLOK1-1)/NBLOK1;\
  for(j=0;j<jblok-1;j++){\
    jb=j*NBLOK1+tx;\
    vecl[jb]=lvec[jb];\
    vecl[jb]=lvec[jb];\
    __syncthreads();\
  }\
  if(j*NBLOK1+tx<MPMAX){\
    jb=j*NBLOK1+tx;\
    vecl[jb]=lvec[jb];\
    vecl[jb]=lvec[jb];\
  }\
  __syncthreads();\
  nb=nvec[bx*mblok+10];\
  jbase=nvec[bx*mblok+11];\
  for(i=0;i<3;i++) nc[i]=0;\
  k=0;\
  i=1;\
  while(nb!=0){\
    j=2-k;\
    nc[j]=nc[j]+nb%2*i;\
    nb=nb/2;\
    j=k+1;\
    k=j%3;\
    if(k==0) i=i*2;\
  }\
  nd=nc[0];\
  nc[0]=nc[1];\
  nc[1]=nc[2];\
  nc[2]=nd;\
  xic=xmin+(nc[0]+0.5)*rb;\
  yic=ymin+(nc[1]+0.5)*rb;\
  zic=zmin+(nc[2]+0.5)*rb;\
  jb=jbase+tx;\
  for(i=0;i<6;i++) veci[6*tx+i]=ivec[6*ib+i];\
  for(i=0;i<2;i++) vecj[2*tx+i]=jvec[2*jb+i];\
  for(i=0;i<4;i++) veck[4*tx+i]=0.0f;\
  __syncthreads();\
  l2p_kernel_core;\
  for(i=0;i<4;i++) kvec[4*ib+i]=veck[4*tx+i];\
}

P2P_KERNEL(G_p2p_kernel,G_P2P_KERNEL_CORE);
P2P_KERNEL(Gni_p2p_kernel,GNI_P2P_KERNEL_CORE);
P2P_KERNEL(Gnj_p2p_kernel,GNJ_P2P_KERNEL_CORE);
P2M_KERNEL(G_p2m_kernel,G_P2M_KERNEL_CORE);
P2M_KERNEL(Gn_p2m_kernel,GN_P2M_KERNEL_CORE);
M2L_KERNEL(m2m_kernel,M2M_KERNEL_CORE);
M2L_KERNEL(m2l_kernel,M2L_KERNEL_CORE);
M2L_KERNEL(l2l_kernel,L2L_KERNEL_CORE);
L2P_KERNEL(G_l2p_kernel,G_L2P_KERNEL_CORE);
L2P_KERNEL(Gn_l2p_kernel,GN_L2P_KERNEL_CORE);
