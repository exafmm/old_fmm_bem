
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 23      1    23
      /   :  6      4    24
      sqrt:  1      4     4
      pow :  1     13    13
      sum                64
*/
#define GNI_P2P_KERNEL_CORE \
      dxij=veci[0]-vecj[jj7  ];\
      dyij=veci[1]-vecj[jj7+1];\
      dzij=veci[2]-vecj[jj7+2];\
      rij=rsqrtf(dxij*dxij+dyij*dyij+dzij*dzij+eps);\
      rsij=pi14*vecj[jj7+6]*rij*rij*rij;\
      veck[0]+=(dxij*veci[3]+dyij*veci[4]+dzij*veci[5])*rsij;\
      jj7+=7;
