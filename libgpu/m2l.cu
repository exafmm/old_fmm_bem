
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 27      1    27
      /   :  2      4     8
      pow :  1     13    13
      sum                48
*/
#define M2L_KERNEL_CORE \
    for(i=0;i<3;i++) nc[i]=0;\
    nb=je-1;\
    k=0;\
    i=1;\
    while(nb!=0){\
      j=2-k;\
      nc[j]=nc[j]+nb%2*i;\
      nb=nb/2;\
      j=k+1;\
      k=j%3;\
      if(k==0) i=i*2;\
    }\
    nd=nc[0];\
    nc[0]=nc[1];\
    nc[1]=nc[2];\
    nc[2]=nd;\
    xijc=(nc[0]-3)*rb;\
    yijc=(nc[1]-3)*rb;\
    zijc=(nc[2]-3)*rb;\
    rh=sqrt(xijc*xijc+yijc*yijc+zijc*zijc)+eps;\
    jbase=(je-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*nms+i]=vecd[i];\
    __syncthreads();\
    j=ng[tx];\
    k=mg[tx];\
    jks=j*(j+1)/2+k;\
    for(i=0;i<2;i++) vecd[i]=0;\
    fnmm=1.0;\
    for(i=0;i<j-k;i++) fnmm=fnmm*(i+1);\
    fnpm=1.0;\
    for(i=0;i<j+k;i++) fnpm=fnpm*(i+1);\
    ajk=pow(-1.0,j)*rsqrtf(fnmm*fnpm);\
    for(n=abs(k);n<mp;n++){\
      nks=n*(n+1)/2+k;\
      jnk=(j+n)*(j+n)+j+n;\
      fnmm=1.0;\
      for(i=0;i<n-k;i++) fnmm=fnmm*(i+1);\
      fnpm=1.0;\
      for(i=0;i<n+k;i++) fnpm=fnpm*(i+1);\
      ank=pow(-1.0,n)*rsqrtf(fnmm*fnpm);\
      fnpm=1.0;\
      for(i=0;i<j+n;i++) fnpm=fnpm*(i+1);\
      ajn=pow(-1.0,j+n)/fnpm;\
      sr=pow(-1.0,j+k)*ank*ajk/ajn;\
      cnmre=sr*ynmre[jnk]/pow(rh,j+n+1);\
      cnmim=sr*ynmim[jnk]/pow(rh,j+n+1);\
      vecd[0]+=vecj[2*nks+0]*cnmre;\
      vecd[0]-=vecj[2*nks+1]*cnmim;\
      vecd[1]+=vecj[2*nks+0]*cnmim;\
      vecd[1]+=vecj[2*nks+1]*cnmre;\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*jks+i]=vecd[i];\
    __syncthreads();\
    jbase=(je+nrbm-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }
