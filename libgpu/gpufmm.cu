#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <cutil.h>
#include <multithreading.h>

#include "var.h"
#include "G_p2p.cu"
#include "Gni_p2p.cu"
#include "Gnj_p2p.cu"
#include "G_p2m.cu"
#include "Gn_p2m.cu"
#include "m2m.cu"
#include "m2l.cu"
#include "l2l.cu"
#include "G_l2p.cu"
#include "Gn_l2p.cu"
#include "kernel.cu"
#include "wrapper.cu"

void G_p2p_gpu();
void Gni_p2p_gpu();
void Gnj_p2p_gpu();
void G_p2m_gpu();
void Gn_p2m_gpu();
void m2m_gpu();
void m2l_gpu();
void l2l_gpu();
void G_l2p_gpu();
void Gn_l2p_gpu();

double get_gpu_time(void)
{
  struct timeval tv;
  struct timezone tz;
  if (is_set==1) hipDeviceSynchronize();
  gettimeofday(&tv, &tz);
  return ((double)(tv.tv_sec+tv.tv_usec*1.0e-6));
}

SS add_nitadori(SS a, float ys)
{
  float ws;
  SS b;
  b.hs=a.hs+ys;
  ws=b.hs-a.hs;
  b.ls=ys-ws;
  b.ls=a.ls+b.ls;
  return b;
}

extern void p2pgpu_(int nvecd[],double *op,
	float xi[],float yi[],float zi[],
	float gxi[],float gyi[],float gzi[],float vi[],
	float xj[],float yj[],float zj[],
	float gxj[],float gyj[],float gzj[],float vj[])
{
  int i,nn,ni,nk,nflop;
  idev=nvecd[0];
  iblok=nvecd[1];
  int mblok=nvecd[2],nj=nvecd[3],neq=nvecd[4],myrank=nvecd[5];
  double tic,toc,flops,t[10];

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();
  nn=iblok*mblok+10;
  ni=iblok*NBLOK0;
  nk=iblok*NBLOK0;
  ms=sizeof(float)*NSCAL;
  mn=sizeof(int)*ROUNDUP0(nn);
  mi=sizeof(float)*6*ROUNDUP0(ni);
  mj=sizeof(float)*7*ROUNDUP0(nj);
  mk=sizeof(float)*4*ROUNDUP0(nk);

  scald=(float *)malloc(ms);
  nvec=(int *)malloc(mn);
  ivec=(float *)malloc(mi);
  jvec=(float *)malloc(mj);
  kvec=(float *)malloc(mk);

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(idev));
    CUDA_SAFE_CALL(hipGetDevice(&jdev));
    assert(idev==jdev);
    for(i=0;i<10;i++) tgpu[i]=0;
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (mn>mn_a) {
    if(mn_a!=0) CUDA_SAFE_CALL(hipFree(nveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(nveg+idev),mn));
    mn_a=mn;
  }
  if (mi>mi_a) {
    if(mi_a!=0) CUDA_SAFE_CALL(hipFree(iveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(iveg+idev),mi));
    mi_a=mi;
  }
  if (mj>mj_a) {
    if(mj_a!=0) CUDA_SAFE_CALL(hipFree(jveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(jveg+idev),mj));
    mj_a=mj;
  }
  if (mk>mk_a) {
    if(mk_a!=0) CUDA_SAFE_CALL(hipFree(kveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(kveg+idev),mk));
    mk_a=mk;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  scald[0]=0;
  scald[1]=0;
  scald[2]=0;
  scald[3]=0;
  scald[4]=0;
  scald[5]=0;
  for(i=0;i<nn;i++){
    nvec[i]=nvecd[i];
  }
  for(i=0;i<ni;i++){
    ivec[6*i+0]=xi[i];
    ivec[6*i+1]=yi[i];
    ivec[6*i+2]=zi[i];
    ivec[6*i+3]=gxi[i];
    ivec[6*i+4]=gyi[i];
    ivec[6*i+5]=gzi[i];
  }
  for(i=0;i<nj;i++){
    jvec[7*i+0]=xj[i];
    jvec[7*i+1]=yj[i];
    jvec[7*i+2]=zj[i];
    jvec[7*i+3]=gxj[i];
    jvec[7*i+4]=gyj[i];
    jvec[7*i+5]=gzj[i];
    jvec[7*i+6]=vj[i];
  }

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(scal),scald,ms));
  CUDA_SAFE_CALL(hipMemcpy(nveg[idev],nvec,mn,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(iveg[idev],ivec,mi,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(jveg[idev],jvec,mj,hipMemcpyHostToDevice));
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  switch(neq){
    case 0 : G_p2p_gpu(); nflop=64 ;break;
    case 1 : Gni_p2p_gpu(); nflop=64 ;break;
    case 2 : Gnj_p2p_gpu(); nflop=64 ;break;
  }

  toc=tic;
  tic=get_gpu_time();
  t[3]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpy(kvec,kveg[idev],mk,hipMemcpyDeviceToHost));
  CUT_THREADEND;
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  for(i=0;i<nk;i++){
     vi[i]=kvec[4*i+0];
     gxi[i]=kvec[4*i+1];
     gyi[i]=kvec[4*i+2];
     gzi[i]=kvec[4*i+3];
  }

  free(scald);
  free(nvec);
  free(ivec);
  free(jvec);
  free(kvec);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  for(i=0;i<10;i++) tgpu[i]+=t[i];
  flops=*op*((double)nflop)/t[9];
//  printf("[%2d:%d] p2p hipMalloc : %f s\n",myrank,idev,t[1]);
//  printf("[%2d:%d] p2p hipMemcpy : %f s\n",myrank,idev,t[2]);
//  printf("[%2d:%d] p2p cudaKernel : %f s\n",myrank,idev,t[3]);
//  printf("[%2d:%d] p2p other      : %f s\n",myrank,idev,t[0]);
//  printf("[%2d:%d] p2p flops      : %f G\n",myrank,idev,flops/1e9);
  tic=flops+myrank;
}

extern void p2mgpu_(int nvecd[],
        double *op,double *rb,
        double *xmin,double *ymin,double *zmin,
        float  xjg[],float  yjg[],float  zjg[],
        float gxjg[],float gyjg[],float gzjg[], float vjg[],
        float brex[],float bimx[],float fac[])
{
  int i,nn,ni,nk,nflop;
  double tic,toc,flops,t[10];

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();
  idev=nvecd[0];
  iblok=nvecd[1];
  int mblok=nvecd[2],nj=nvecd[3],neq=nvecd[4],myrank=nvecd[5],mp=nvecd[6];
  nn=iblok*mblok+10;
  ni=iblok*NBLOK1;
  nk=mp*mp;
  ms=sizeof(float)*NSCAL;
  mn=sizeof(int)*ROUNDUP1(nn);
  mi=sizeof(float)*2*ROUNDUP1(ni);
  mj=sizeof(float)*7*ROUNDUP1(nj);
  mk=sizeof(float)*ROUNDUP1(nk);

  scald=(float *)malloc(ms);
  nvec=(int *)malloc(mn);
  ivec=(float *)malloc(mi);
  jvec=(float *)malloc(mj);
  kvec=(float *)malloc(mk);

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(idev));
    CUDA_SAFE_CALL(hipGetDevice(&jdev));
    assert(idev==jdev);
    for(i=0;i<10;i++) tgpu[i]=0;
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (mn>mn_a) {
    if(mn_a!=0) CUDA_SAFE_CALL(hipFree(nveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(nveg+idev),mn));
    mn_a=mn;
  }
  if (mi>mi_a) {
    if(mi_a!=0) CUDA_SAFE_CALL(hipFree(iveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(iveg+idev),mi));
    mi_a=mi;
  }
  if (mj>mj_a) {
    if(mj_a!=0) CUDA_SAFE_CALL(hipFree(jveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(jveg+idev),mj));
    mj_a=mj;
  }
  if (mk>mk_a) {
    if(mk_a!=0) CUDA_SAFE_CALL(hipFree(kveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(kveg+idev),mk));
    mk_a=mk;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  scald[0]=(float)*rb;
  scald[1]=(float)*xmin;
  scald[2]=(float)*ymin;
  scald[3]=(float)*zmin;
  scald[4]=0;
  scald[5]=0;
  for(i=0;i<nn;i++){
    nvec[i]=nvecd[i];
  }
  for(i=0;i<nj;i++){
    jvec[7*i+0]= xjg[i];
    jvec[7*i+1]= yjg[i];
    jvec[7*i+2]= zjg[i];
    jvec[7*i+3]=gxjg[i];
    jvec[7*i+4]=gyjg[i];
    jvec[7*i+5]=gzjg[i];
    jvec[7*i+6]= vjg[i];
  }
  for(i=0;i<nk;i++){
    kvec[i]=fac[i];
  }

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(scal),scald,ms));
  CUDA_SAFE_CALL(hipMemcpy(nveg[idev],nvec,mn,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(jveg[idev],jvec,mj,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(kveg[idev],kvec,mk,hipMemcpyHostToDevice));
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  switch(neq){
    case 0 : G_p2m_gpu(); nflop=291 ;break;
    case 1 : Gn_p2m_gpu(); nflop=568 ;break;
  }

  toc=tic;
  tic=get_gpu_time();
  t[3]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpy(ivec,iveg[idev],mi,hipMemcpyDeviceToHost));
  CUT_THREADEND;
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  for(i=0;i<ni;i++){
    brex[i]=ivec[2*i+0];
    bimx[i]=ivec[2*i+1];
  }

  free(scald);
  free(nvec);
  free(ivec);
  free(jvec);
  free(kvec);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  for(i=0;i<10;i++) tgpu[i]+=t[i];
  flops=*op*((double)nflop)/t[9];
//  printf("[%2d:%d] p2m hipMalloc : %f s\n",myrank,idev,t[1]);
//  printf("[%2d:%d] p2m hipMemcpy : %f s\n",myrank,idev,t[2]);
//  printf("[%2d:%d] p2m cudaKernel : %f s\n",myrank,idev,t[3]);
//  printf("[%2d:%d] p2m other      : %f s\n",myrank,idev,t[0]);
//  printf("[%2d:%d] p2m flops      : %f G\n",myrank,idev,flops/1e9);
  tic=flops+myrank;
}

extern void m2lgpu_(int nvecd[],double *op,double *rb,
	float arex[],float aimx[],float brex[],float bimx[],
	float ynmre[],float ynmim[],float dnmre[],float dnmim[])
{
  int i,nn,ni,nk,nl,nflop;
  double tic,toc,flops,t[10];

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();
  idev=nvecd[0];
  iblok=nvecd[1];
  int mblok=nvecd[2],nj=nvecd[3],neq=nvecd[4],myrank=nvecd[5],mp=nvecd[6],nrbm=nvecd[7];
  int mpdnm=(4*mp*mp*mp-mp)/3;
  nn=iblok*mblok+10;
  ni=iblok*NBLOK1;
  nk=mp*mp;
  nl=mpdnm*2*nrbm;
  ms=sizeof(float)*NSCAL;
  mn=sizeof(int)*ROUNDUP1(nn);
  mi=sizeof(float)*2*ROUNDUP1(ni);
  mj=sizeof(float)*2*ROUNDUP1(nj);
  mk=sizeof(float)*2*ROUNDUP1(nk);
  ml=sizeof(float)*2*ROUNDUP1(nl);

  scald=(float *)malloc(ms);
  nvec=(int *)malloc(mn);
  ivec=(float *)malloc(mi);
  jvec=(float *)malloc(mj);
  kvec=(float *)malloc(mk);
  lvec=(float *)malloc(ml);

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(idev));
    CUDA_SAFE_CALL(hipGetDevice(&jdev));
    assert(idev==jdev);
    for(i=0;i<10;i++) tgpu[i]=0;
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (mn>mn_a) {
    if(mn_a!=0) CUDA_SAFE_CALL(hipFree(nveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(nveg+idev),mn));
    mn_a=mn;
  }
  if (mi>mi_a) {
    if(mi_a!=0) CUDA_SAFE_CALL(hipFree(iveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(iveg+idev),mi));
    mi_a=mi;
  }
  if (mj>mj_a) {
    if(mj_a!=0) CUDA_SAFE_CALL(hipFree(jveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(jveg+idev),mj));
    mj_a=mj;
  }
  if (mk>mk_a) {
    if(mk_a!=0) CUDA_SAFE_CALL(hipFree(kveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(kveg+idev),mk));
    mk_a=mk;
  }
  if (ml>ml_a) {
    if(ml_a!=0) CUDA_SAFE_CALL(hipFree(lveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(lveg+idev),ml));
    ml_a=ml;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  scald[0]=(float)*rb;
  scald[1]=0;
  scald[2]=0;
  scald[3]=0;
  scald[4]=0;
  scald[5]=0;
  for(i=0;i<nn;i++){
    nvec[i]=nvecd[i];
  }
  for(i=0;i<nj;i++){
    jvec[2*i+0]=brex[i];
    jvec[2*i+1]=bimx[i];
  }
  for(i=0;i<nk;i++){
    kvec[2*i+0]=ynmre[i];
    kvec[2*i+1]=ynmim[i];
  }
  for(i=0;i<nl;i++){
    lvec[2*i+0]=dnmre[i];
    lvec[2*i+1]=dnmim[i];
  }

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(scal),scald,ms));
  CUDA_SAFE_CALL(hipMemcpy(nveg[idev],nvec,mn,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(jveg[idev],jvec,mj,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(kveg[idev],kvec,mk,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(lveg[idev],lvec,ml,hipMemcpyHostToDevice));
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  switch(neq){
    case 0 : m2m_gpu(); nflop=48 ;break;
    case 1 : m2l_gpu(); nflop=48 ;break;
    case 2 : l2l_gpu(); nflop=48 ;break;
  }

  toc=tic;
  tic=get_gpu_time();
  t[3]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpy(ivec,iveg[idev],mi,hipMemcpyDeviceToHost));
  CUT_THREADEND;
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  for(i=0;i<ni;i++){
    arex[i]=ivec[2*i+0];
    aimx[i]=ivec[2*i+1];
  }

  free(scald);
  free(nvec);
  free(ivec);
  free(jvec);
  free(kvec);
  free(lvec);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  for(i=0;i<10;i++) tgpu[i]+=t[i];
  flops=*op*((double)nflop)/t[9];
//  printf("[%2d:%d] m2l hipMalloc : %f s\n",myrank,idev,t[1]);
//  printf("[%2d:%d] m2l hipMemcpy : %f s\n",myrank,idev,t[2]);
//  printf("[%2d:%d] m2l cudaKernel : %f s\n",myrank,idev,t[3]);
//  printf("[%2d:%d] m2l other      : %f s\n",myrank,idev,t[0]);
//  printf("[%2d:%d] m2l flops      : %f G\n",myrank,idev,flops/1e9);
  tic=flops+myrank;
}

extern void l2pgpu_(int nvecd[],
        double *op,double *rb,
        double *xmin,double *ymin,double *zmin,
        float  xig[],float  yig[],float  zig[],
        float gxig[],float gyig[],float gzig[],float vig[],
        float arex[],float aimx[],float fac[])
{
  int i,nn,ni,nk,nl,nflop;
  double tic,toc,flops,t[10];

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();
  idev=nvecd[0];
  iblok=nvecd[1];
  int mblok=nvecd[2],nj=nvecd[3],neq=nvecd[4],myrank=nvecd[5],mp=nvecd[6];
  nn=iblok*mblok+10;
  ni=iblok*NBLOK1;
  nk=iblok*NBLOK1;
  nl=mp*mp;
  ms=sizeof(float)*NSCAL;
  mn=sizeof(int)*ROUNDUP1(nn);
  mi=sizeof(float)*6*ROUNDUP1(ni);
  mj=sizeof(float)*2*ROUNDUP1(nj);
  mk=sizeof(float)*4*ROUNDUP1(nk);
  ml=sizeof(float)*ROUNDUP1(nl);

  scald=(float *)malloc(ms);
  nvec=(int *)malloc(mn);
  ivec=(float *)malloc(mi);
  jvec=(float *)malloc(mj);
  kvec=(float *)malloc(mk);
  lvec=(float *)malloc(ml);

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(idev));
    CUDA_SAFE_CALL(hipGetDevice(&jdev));
    assert(idev==jdev);
    for(i=0;i<10;i++) tgpu[i]=0;
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (mn>mn_a) {
    if(mn_a!=0) CUDA_SAFE_CALL(hipFree(nveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(nveg+idev),mn));
    mn_a=mn;   
  }
  if (mi>mi_a) {
    if(mi_a!=0) CUDA_SAFE_CALL(hipFree(iveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(iveg+idev),mi));
    mi_a=mi;
  }
  if (mj>mj_a) {
    if(mj_a!=0) CUDA_SAFE_CALL(hipFree(jveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(jveg+idev),mj));
    mj_a=mj;
  }
  if (mk>mk_a) {
    if(mk_a!=0) CUDA_SAFE_CALL(hipFree(kveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(kveg+idev),mk));
    mk_a=mk;
  } 
  if (ml>ml_a) {
    if(ml_a!=0) CUDA_SAFE_CALL(hipFree(lveg[idev]));
    CUDA_SAFE_CALL(hipMalloc((void**)(lveg+idev),ml));
    ml_a=ml;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  scald[0]=(float)*rb;
  scald[1]=(float)*xmin;
  scald[2]=(float)*ymin;
  scald[3]=(float)*zmin;
  scald[4]=0;
  scald[5]=0;
  for(i=0;i<nn;i++){
    nvec[i]=nvecd[i];
  }
  for(i=0;i<ni;i++){
    ivec[6*i+0]= xig[i];
    ivec[6*i+1]= yig[i];
    ivec[6*i+2]= zig[i];
    ivec[6*i+3]=gxig[i];
    ivec[6*i+4]=gyig[i];
    ivec[6*i+5]=gzig[i];
  }
  for(i=0;i<nj;i++){
    jvec[2*i+0]=arex[i];
    jvec[2*i+1]=aimx[i];
  }
  for(i=0;i<nl;i++){
    lvec[i]=fac[i];
  }

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(scal),scald,ms));
  CUDA_SAFE_CALL(hipMemcpy(nveg[idev],nvec,mn,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(iveg[idev],ivec,mi,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(jveg[idev],jvec,mj,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(lveg[idev],lvec,ml,hipMemcpyHostToDevice));
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  switch(neq){
    case 0 : G_l2p_gpu(); nflop=169 ;break;
    case 1 : Gn_l2p_gpu(); nflop=186 ;break;
  }

  toc=tic;
  tic=get_gpu_time();
  t[3]+=tic-toc;
  CUDA_SAFE_CALL(hipMemcpy(kvec,kveg[idev],mk,hipMemcpyDeviceToHost));
  CUT_THREADEND;
  toc=tic;
  tic=get_gpu_time();
  t[2]+=tic-toc;

  for(i=0;i<nk;i++){
     gxig[i]=kvec[4*i+0];
     gyig[i]=kvec[4*i+1];
     gzig[i]=kvec[4*i+2];
     vig[i]=kvec[4*i+3];
  }

  free(scald);
  free(nvec);
  free(ivec);
  free(jvec);
  free(kvec);
  free(lvec);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  for(i=0;i<10;i++) tgpu[i]+=t[i];
  flops=*op*((double)nflop)/t[9];
//  printf("[%2d:%d] l2p hipMalloc : %f s\n",myrank,idev,t[1]);
//  printf("[%2d:%d] l2p hipMemcpy : %f s\n",myrank,idev,t[2]);
//  printf("[%2d:%d] l2p cudaKernel : %f s\n",myrank,idev,t[3]);
//  printf("[%2d:%d] l2p other      : %f s\n",myrank,idev,t[0]);
//  printf("[%2d:%d] l2p flops      : %f G\n",myrank,idev,flops/1e9);
//  printf("[%2d:%d] l2p hipMalloc : %f s\n",myrank,idev,tgpu[1]);
//  printf("[%2d:%d] l2p hipMemcpy : %f s\n",myrank,idev,tgpu[2]);
//  printf("[%2d:%d] l2p cudaKernel : %f s\n",myrank,idev,tgpu[3]);
//  printf("[%2d:%d] l2p other      : %f s\n",myrank,idev,tgpu[0]);
  tic=flops+myrank;
}
