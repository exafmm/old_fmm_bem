
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 27      1    27
      /   :  2      4     8
      pow :  1     13    13
      sum                48
*/
#define L2L_KERNEL_CORE \
    rh=rb*sqrtf(3.0f)/2;\
    jbase=(je-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*nms+i]=vecd[i];\
    __syncthreads();\
    j=ng[tx];\
    k=mg[tx];\
    jks=j*(j+1)/2+k;\
    for(i=0;i<2;i++) vecd[i]=0;\
    fnmm=1.0;\
    for(i=0;i<j-k;i++) fnmm=fnmm*(i+1);\
    fnpm=1.0;\
    for(i=0;i<j+k;i++) fnpm=fnpm*(i+1);\
    ajk=pow(-1.0,j)*rsqrtf(fnmm*fnpm);\
    for(n=j;n<mp;n++){\
      nks=n*(n+1)/2+k;\
      jnk=(n-j)*(n-j)+n-j;\
      fnmm=1.0;\
      for(i=0;i<n-k;i++) fnmm=fnmm*(i+1);\
      fnpm=1.0;\
      for(i=0;i<n+k;i++) fnpm=fnpm*(i+1);\
      ank=pow(-1.0,n)*rsqrtf(fnmm*fnpm);\
      fnpm=1.0;\
      for(i=0;i<n-j;i++) fnpm=fnpm*(i+1);\
      ajn=pow(-1.0,n-j)/fnpm;\
      sr=ajn*ajk/ank;\
      cnmre=sr*ynmre[jnk]*pow(rh,n-j);\
      cnmim=sr*ynmim[jnk]*pow(rh,n-j);\
      vecd[0]+=vecj[2*nks+0]*cnmre;\
      vecd[0]-=vecj[2*nks+1]*cnmim;\
      vecd[1]+=vecj[2*nks+0]*cnmim;\
      vecd[1]+=vecj[2*nks+1]*cnmre;\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*jks+i]=vecd[i];\
    __syncthreads();\
    jbase=(je+nrbm-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }
