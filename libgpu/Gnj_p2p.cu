
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 23      1    23
      /   :  6      4    24
      sqrt:  1      4     4
      pow :  1     13    13
      sum                64
*/
#define GNJ_P2P_KERNEL_CORE \
      dxij=veci[0]-vecj[jj7  ];\
      dyij=veci[1]-vecj[jj7+1];\
      dzij=veci[2]-vecj[jj7+2];\
      rij=rsqrtf(dxij*dxij+dyij*dyij+dzij*dzij+eps);\
      rsij=pi14*vecj[jj7+6]*rij*rij*rij;\
      veck[0]+=(dxij*vecj[jj7+3]+dyij*vecj[jj7+4]+dzij*vecj[jj7+5])*rsij;\
      jj7+=7;
