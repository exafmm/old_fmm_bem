
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* :326      1   326
      /   : 40      4   160
      sqrt:  2      4     8
      sin :  3      8    24
      cos :  3      8    24
      pow :  2     13    26
      sum               568
*/
#define GN_P2M_KERNEL_CORE \
      float yy;\
      float brerh,breal,brebe,bimrh,bimal,bimbe;\
      float brexd,breyd,brezd,bimxd,bimyd,bimzd;\
      __shared__ float bth[MPMAX];\
      xjjc=vecj[jj7+0]-xjc;\
      yjjc=vecj[jj7+1]-yjc;\
      zjjc=vecj[jj7+2]-zjc;\
      rh=sqrtf(xjjc*xjjc+yjjc*yjjc+zjjc*zjjc)+eps;\
      al=acosf(zjjc/rh);\
      if(abs(xjjc)+abs(yjjc)<eps){\
        be=0;\
      }\
      else if(abs(xjjc)<eps){\
        be=yjjc/abs(yjjc)*M_PI*0.5;\
      }\
      else if(xjjc>0){\
        be=atanf(yjjc/xjjc);\
      }\
      else{\
        be=atanf(yjjc/xjjc)+M_PI;\
      }\
      xx=__cosf(al);\
      yy=__sinf(al);\
      if(fabs(yy)<eps) yy=1/eps;\
      s2=sqrtf((1-xx)*(1+xx));\
      fact=1;\
      pn=1;\
      for(m=0;m<=mg[tx];m++){\
        p=pn;\
        nm=m*m+2*m;\
        bnm[nm]=veck[nm]*p;\
        p1=p;\
        p=xx*(2*m+1)*p;\
        bth[nm]=veck[nm]*(p-(m+1)*xx*p1)/yy;\
        for(n=m+1;n<=ng[tx];n++){\
          nm=n*n+n+m;\
          bnm[nm]=veck[nm]*p;\
          p2=p1;\
          p1=p;\
          p=(xx*(2*n+1)*p1-(n+m)*p2)/(n-m+1);\
          bth[nm]=veck[nm]*((n-m+1)*p-(n+1)*xx*p1)/yy;\
        }\
        pn=-pn*fact*s2;\
        fact=fact+2;\
      }\
      n=ng[tx];\
      m=mg[tx];\
      nms=n*(n+1)/2+m;\
      nm=n*n+n+m;\
      ere=__cosf(-m*be);\
      eim=__sinf(-m*be);\
      rhm=powf(rh,n-1);\
      rhn=rhm*rh;\
      brerh=n*rhm*bnm[nm]*ere;\
      breal=rhn*bth[nm]*ere;\
      brebe=m*rhn*bnm[nm]*eim;\
      bimrh=n*rhm*bnm[nm]*eim;\
      bimal=rhn*bth[nm]*eim;\
      bimbe=-m*rhn*bnm[nm]*ere;\
      brexd=__sinf(al)*__cosf(be)*brerh+__cosf(al)*__cosf(be)/rh*breal-\
      __sinf(be)/rh/yy*brebe;\
      breyd=__sinf(al)*__sinf(be)*brerh+__cosf(al)*__sinf(be)/rh*breal+\
      __cosf(be)/rh/yy*brebe;\
      brezd=__cosf(al)*brerh-__sinf(al)/rh*breal;\
      bimxd=__sinf(al)*__cosf(be)*bimrh+__cosf(al)*__cosf(be)/rh*bimal-\
      __sinf(be)/rh/yy*bimbe;\
      bimyd=__sinf(al)*__sinf(be)*bimrh+__cosf(al)*__sinf(be)/rh*bimal+\
      __cosf(be)/rh/yy*bimbe;\
      bimzd=__cosf(al)*bimrh-__sinf(al)/rh*bimal;\
      veci[2*tx+0]+=vecj[jj7+6]*vecj[jj7+3]*brexd;\
      veci[2*tx+0]+=vecj[jj7+6]*vecj[jj7+4]*breyd;\
      veci[2*tx+0]+=vecj[jj7+6]*vecj[jj7+5]*brezd;\
      veci[2*tx+1]+=vecj[jj7+6]*vecj[jj7+3]*bimxd;\
      veci[2*tx+1]+=vecj[jj7+6]*vecj[jj7+4]*bimyd;\
      veci[2*tx+1]+=vecj[jj7+6]*vecj[jj7+5]*bimzd;\
      jj7+=7;
