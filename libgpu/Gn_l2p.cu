
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 37      1    37
      /   :  3      4    12
      sin :  1      8     8
      cos :  1      8     8
      pow :  8     13   104
      sum               169
*/
#define GN_L2P_KERNEL_CORE \
      xiic=veci[6*tx+0]-xic;\
      yiic=veci[6*tx+1]-yic;\
      ziic=veci[6*tx+2]-zic;\
      r=sqrtf(xiic*xiic+yiic*yiic+ziic*ziic)+eps;\
      th=acosf(ziic/r);\
      if(abs(xiic)+abs(yiic)<eps){\
        ph=0;\
      }\
      else if(abs(xiic)<eps){\
        ph=yiic/abs(yiic)*M_PI*0.5;\
      }\
      else if(xiic>0){\
        ph=atanf(yiic/xiic);\
      }\
      else{\
        ph=atanf(yiic/xiic)+M_PI;\
      }\
      gr=0;\
      gth=0;\
      gph=0;\
      xx=__cosf(th);\
      yy=__sinf(th);\
      if(fabs(yy)<eps) yy=1/eps;\
      s2=sqrtf((1-xx)*(1+xx));\
      fact=1;\
      pn=1;\
      for(m=0;m<mp;m++){\
        p=pn;\
        nm=m*m+2*m;\
        nms=m*(m+1)/2+m;\
        ere=__cosf(m*ph);\
        if(m==0) ere=0.5;\
        eim=__sinf(m*ph);\
        bnm=vecl[nm]*p;\
        p1=p;\
        p=xx*(2*m+1)*p;\
        bth=vecl[nm]*(p-(m+1)*xx*p1)/yy;\
        rrre=m*powf(r,m-1)*bnm*ere;\
        rthre=powf(r,m)*bth*ere;\
        rphre=-m*powf(r,m)*bnm*eim;\
        rrim=m*powf(r,m-1)*bnm*eim;\
        rthim=powf(r,m)*bth*eim;\
        rphim=m*powf(r,m)*bnm*ere;\
        gr+=2*(rrre*vecj[2*nms+0]-rrim*vecj[2*nms+1]);\
        gth+=2*(rthre*vecj[2*nms+0]-rthim*vecj[2*nms+1]);\
        gph+=2*(rphre*vecj[2*nms+0]-rphim*vecj[2*nms+1]);\
        for(n=m+1;n<mp;n++){\
          nm=n*n+n+m;\
          nms=n*(n+1)/2+m;\
          bnm=vecl[nm]*p;\
          p2=p1;\
          p1=p;\
          p=(xx*(2*n+1)*p1-(n+m)*p2)/(n-m+1);\
          bth=vecl[nm]*((n-m+1)*p-(n+1)*xx*p1)/yy;\
          rrre=n*powf(r,n-1)*bnm*ere;\
          rthre=powf(r,n)*bth*ere;\
          rphre=-m*powf(r,n)*bnm*eim;\
          rrim=n*powf(r,n-1)*bnm*eim;\
          rthim=powf(r,n)*bth*eim;\
          rphim=m*powf(r,n)*bnm*ere;\
          gr+=2*(rrre*vecj[2*nms+0]-rrim*vecj[2*nms+1]);\
          gth+=2*(rthre*vecj[2*nms+0]-rthim*vecj[2*nms+1]);\
          gph+=2*(rphre*vecj[2*nms+0]-rphim*vecj[2*nms+1]);\
        }\
        pn=-pn*fact*s2;\
        fact=fact+2;\
      }\
      gx=__sinf(th)*__cosf(ph)*gr+__cosf(th)*__cosf(ph)/r*gth-\
        __sinf(ph)/r/yy*gph;\
      gy=__sinf(th)*__sinf(ph)*gr+__cosf(th)*__sinf(ph)/r*gth+\
        __cosf(ph)/r/yy*gph;\
      gz=__cosf(th)*gr-__sinf(th)/r*gth;\
      veck[tx]-=0.25/M_PI*(gx*veci[6*tx+3]+gy*veci[6*tx+4]+gz*veci[6*tx+5]);
