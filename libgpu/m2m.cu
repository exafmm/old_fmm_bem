
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 27      1    27
      /   :  2      4     8
      pow :  1     13    13
      sum                48
*/
#define M2M_KERNEL_CORE \
    rh=rb*sqrtf(3.0f)/4;\
    jbase=(je-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*nms+i]=vecd[i];\
    __syncthreads();\
    j=ng[tx];\
    k=mg[tx];\
    jks=j*(j+1)/2+k;\
    for(i=0;i<2;i++) vecd[i]=0;\
    fnmm=1.0;\
    for(i=0;i<j-k;i++) fnmm=fnmm*(i+1);\
    fnpm=1.0;\
    for(i=0;i<j+k;i++) fnpm=fnpm*(i+1);\
    ajk=pow(-1.0,j)*rsqrtf(fnmm*fnpm);\
    for(n=0;n<=j-abs(k);n++){\
      nks=(j-n)*(j-n+1)/2+k;\
      jnk=n*n+n;\
      fnmm=1.0;\
      for(i=0;i<j-n-k;i++) fnmm=fnmm*(i+1);\
      fnpm=1.0;\
      for(i=0;i<j-n+k;i++) fnpm=fnpm*(i+1);\
      ank=pow(-1.0,j-n)*rsqrtf(fnmm*fnpm);\
      fnpm=1.0;\
      for(i=0;i<n;i++) fnpm=fnpm*(i+1);\
      ajn=pow(-1.0,n)/fnpm;\
      sr=pow(-1.0,n)*ank*ajn/ajk;\
      cnmre=sr*ynmre[jnk]*pow(rh,n);\
      cnmim=sr*ynmim[jnk]*pow(rh,n);\
      vecd[0]+=vecj[2*nks+0]*cnmre;\
      vecd[0]-=vecj[2*nks+1]*cnmim;\
      vecd[1]+=vecj[2*nks+0]*cnmim;\
      vecd[1]+=vecj[2*nks+1]*cnmre;\
    }\
    __syncthreads();\
    for(i=0;i<2;i++) vecj[2*jks+i]=vecd[i];\
    __syncthreads();\
    jbase=(je+nrbm-1)*mpdnm;\
    n=ng[tx];\
    m=mg[tx];\
    nms=n*(n+1)/2+m;\
    for(i=0;i<2;i++) vecd[i]=0;\
    for(k=-n;k<0;k++){\
      nks=n*(n+1)/2-k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]+=dnmim*vecj[2*nks+1];\
      vecd[1]-=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }\
    for(k=0;k<=n;k++){\
      nks=n*(n+1)/2+k;\
      nmk=jbase+(4*n*n*n+6*n*n+5*n)/3+m*(2*n+1)+k;\
      dnmre=lvec[2*nmk+0];\
      dnmim=lvec[2*nmk+1];\
      vecd[0]+=dnmre*vecj[2*nks+0];\
      vecd[0]-=dnmim*vecj[2*nks+1];\
      vecd[1]+=dnmre*vecj[2*nks+1];\
      vecd[1]+=dnmim*vecj[2*nks+0];\
    }
