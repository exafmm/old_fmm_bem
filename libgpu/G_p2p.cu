
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* : 23      1    23
      /   :  6      4    24
      sqrt:  1      4     4
      pow :  1     13    13
      sum                64
*/
#define G_P2P_KERNEL_CORE \
      dxij=veci[0]-vecj[jj7  ];\
      dyij=veci[1]-vecj[jj7+1];\
      dzij=veci[2]-vecj[jj7+2];\
      rij=rsqrtf(dxij*dxij+dyij*dyij+dzij*dzij+eps);\
      rsij=pi14*vecj[jj7+6]*rij*rij*rij;\
      veck[0]+=pi14*vecj[jj7+6]*rij;\
      veck[1]-=dxij*rsij;\
      veck[2]-=dyij*rsij;\
      veck[3]-=dzij*rsij;\
      jj7+=7;
