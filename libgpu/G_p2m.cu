
#include <hip/hip_runtime.h>
/*
number of mathematical operations (only floating point)
      operation  flo/o total
      +-* :190      1   190
      /   : 12      4    48
      sqrt:  2      4     8
      sin :  1      8     8
      cos :  3      8    24
      pow :  1     13    13
      sum               291
*/
#define G_P2M_KERNEL_CORE \
      xjjc=vecj[jj7+0]-xjc;\
      yjjc=vecj[jj7+1]-yjc;\
      zjjc=vecj[jj7+2]-zjc;\
      rh=sqrtf(xjjc*xjjc+yjjc*yjjc+zjjc*zjjc)+eps;\
      al=acosf(zjjc/rh);\
      if(abs(xjjc)+abs(yjjc)<eps){\
        be=0;\
      }\
      else if(abs(xjjc)<eps){\
        be=yjjc/abs(yjjc)*M_PI*0.5;\
      }\
      else if(xjjc>0){\
        be=atanf(yjjc/xjjc);\
      }\
      else{\
        be=atanf(yjjc/xjjc)+M_PI;\
      }\
      xx=__cosf(al);\
      s2=sqrtf((1-xx)*(1+xx));\
      fact=1;\
      pn=1;\
      rhm=1;\
      for(m=0;m<=mg[tx];m++){\
        p=pn;\
        nm=m*m+2*m;\
        bnm[nm]=rhm*veck[nm]*p;\
        p1=p;\
        p=xx*(2*m+1)*p;\
        rhm*=rh;\
        rhn=rhm;\
        for(n=m+1;n<=ng[tx];n++){\
          nm=n*n+n+m;\
          bnm[nm]=rhn*veck[nm]*p;\
          p2=p1;\
          p1=p;\
          p=(xx*(2*n+1)*p1-(n+m)*p2)/(n-m+1);\
          rhn*=rh;\
        }\
        pn=-pn*fact*s2;\
        fact=fact+2;\
      }\
      n=ng[tx];\
      m=mg[tx];\
      nm=n*n+n+m;\
      ere=__cosf(-m*be);\
      eim=__sinf(-m*be);\
      veci[2*tx+0]+=vecj[jj7+6]*bnm[nm]*ere;\
      veci[2*tx+1]+=vecj[jj7+6]*bnm[nm]*eim;\
      jj7+=7;
