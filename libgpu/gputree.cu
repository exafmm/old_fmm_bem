#include "hip/hip_runtime.h"
// Copyright(C) 2010 by 
// Tsuyoshi Hamada <hamada@progrape.jp>
// Keigo Nitadori <nitadori@margaux.astron.s.u-tokyo.ac.jp>
// Rio Yokota <rio.yokota@bristol.ac.uk>

#include "vforce.h"

namespace libcunbody{
using namespace std;

__device__ float4 dev_apot(float4 xi, float4 xj, float4 apot)
{
  float dx = xj.x - xi.x;
  float dy = xj.y - xi.y;
  float dz = xj.z - xi.z;
  float eps2 = xi.w;
  float mj   = xj.w;
  float r2   = ((eps2 + dx*dx) + dy*dy) + dz*dz;
  float r1i  = rsqrt(r2);
  float r2i  = r1i*r1i;
  float mr1i = mj * r1i;
  float mr3i = mr1i * r2i;
  apot.x += dx * mr3i;
  apot.y += dy * mr3i;
  apot.z += dz * mr3i;
  apot.w -= mr1i;
  return (apot);
}

__global__ void cunbody_kernel_tree_015(
  float4 *xilist, 
  float4 *xjlist, 
  float4 *apotlist, 
  int2 *off)
{
  const int NJBLOCK = 128;
  int gid = threadIdx.x + blockDim.x * blockIdx.x;
  float4 xi = xilist[gid];
  float4 apot = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    
  int jstart = off[gid].x;
  int jend   = off[gid].y;

  for(int jbase=jstart; jbase<jend; jbase+=NJBLOCK){
#pragma unroll 128
    for(int j=0; j<NJBLOCK; j++){
      float4 xj = xjlist[jbase + j];
      apot = dev_apot (xi, xj, apot);
    }
  }
  apotlist[gid] = apot;
}

}; // namespace libcunbody __END__
